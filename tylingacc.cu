#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void image_convolution_kernel(const unsigned char *input, unsigned char *output, int width, int height, int channels, const float *kernel, int kernel_size)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int tile_x = bx * TILE_SIZE + tx;
    int tile_y = by * TILE_SIZE + ty;
    int c = bz;

    __shared__ float tile[TILE_SIZE + 2][TILE_SIZE + 2];

    // Load data into the shared memory tile
    #pragma acc loop gang worker vector(32)
    for (int ky = -1; ky <= TILE_SIZE; ky += TILE_SIZE)
    {
        #pragma acc loop gang worker vector(32)
        for (int kx = -1; kx <= TILE_SIZE; kx += TILE_SIZE)
        {
            int ix = tile_x + kx;
            int iy = tile_y + ky;

            // Apply boundary conditions
            ix = max(0, min(width - 1, ix));
            iy = max(0, min(height - 1, iy));

            tile[ty + ky + 1][tx + kx + 1] = input[(iy * width + ix) * channels + c] / 255.0f;
        }
    }

    #pragma acc barrier

    int pad = kernel_size / 2;
    float sum = 0;

    #pragma acc loop gang worker vector(32)
    for (int ky = 0; ky < kernel_size; ++ky)
    {
        #pragma acc loop gang worker vector(32)
        for (int kx = 0; kx < kernel_size; ++kx)
        {
            int ix = tx + kx;
            int iy = ty + ky;

            float pixel = tile[iy][ix];
            float kernel_val = kernel[ky * kernel_size + kx];
            sum += pixel * kernel_val;
        }
    }

    // Write the result back to the output
    output[(tile_y * width + tile_x) * channels + c] = (unsigned char)(sum * 255);
}

int main(int argc, char *argv[])
{
    const char *input_file = "input.jpg";
    const char *output_file = "output.jpg";
    int width, height, channels;

    unsigned char *image_data = stbi_load(input_file, &width, &height, &channels, 0);
    if (!image_data)
    {
        fprintf(stderr, "Error loading image\n");
        return 1;
    }

    float kernel[] = {
        -2, -1, 0,
        -1, 1, 1,
        0, 1, 2};
    int kernel_size = 3;

    unsigned char *d_input, *d_output;
    float *d_kernel;

    size_t image_size = width * height * channels * sizeof(unsigned char);

    hipMalloc((void **)&d_input, image_size);
    hipMalloc((void **)&d_output, image_size);
    hipMalloc((void **)&d_kernel, kernel_size * kernel_size * sizeof(float));

    hipMemcpy(d_input, image_data, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((width + TILE_SIZE - 1) / TILE_SIZE,
                   (height + TILE_SIZE - 1) / TILE_SIZE,
                   channels);

    #pragma acc data copyin(d_input[0:image_size], d_kernel[0:kernel_size * kernel_size]) copyout(d_output[0:image_size])
    {
        #pragma acc host_data use_device(d_input, d_output, d_kernel)
        {
            image_convolution_kernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width, height, channels, d_kernel, kernel_size);
        }
    }

    hipMemcpy(image_data, d_output, image_size, hipMemcpyDeviceToHost);

    stbi_write_jpg(output_file, width, height, channels, image_data, 100);

    stbi_image_free(image_data);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);

    return 0;
}
